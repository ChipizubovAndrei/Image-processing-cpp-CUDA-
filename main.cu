#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <stdio.h>
#include <stdint.h>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#define STB_IMAGE_IMPLEMENTATION
#include "lib/stb/stb_image.h"
#include "lib/stb/stb_image_write.h"

#define N_CHANNELS 3
#define MAX_BLOCK_PER_GRID 65534

#include "include/functions.hpp"

int main()
{

  char path_to_image [] = "../images/airplane2.jpg";
  char path_to_save [] = "../output/output2.jpg";

  int count;
  hipDeviceProp_t prop;

  hipGetDeviceCount( &count );

  for (int i = 0; i < count; i++)
  {
    hipGetDeviceProperties( &prop, i );
  }

  hipEvent_t start, stop;
  float gpuTime = 0.0;
  int n_iter = 10;

  uint8_t* src;
  uint8_t* dst;

  uint8_t* dev_src;
  uint8_t* dev_dst;
  uint8_t* dev_kernel;
  uint8_t* dev_transfer_out;

  int width, height, ch;
  uint8_t kernel [81] = {1, 1, 1, 1, 1, 1, 1, 1, 1, 
                        1, 1, 1, 1, 1, 1, 1, 1, 1, 
                        1, 1, 1, 1, 1, 1, 1, 1, 1,
                        1, 1, 1, 1, 1, 1, 1, 1, 1,
                        1, 1, 1, 1, 1, 1, 1, 1, 1,
                        1, 1, 1, 1, 1, 1, 1, 1, 1,
                        1, 1, 1, 1, 1, 1, 1, 1, 1,
                        1, 1, 1, 1, 1, 1, 1, 1, 1,
                        1, 1, 1, 1, 1, 1, 1, 1, 1};

  src = stbi_load( path_to_image, &width, &height, &ch, 3 );

  int N = width*height*N_CHANNELS;
  dst = (uint8_t*)malloc(N * sizeof( uint8_t ));

  hipEventCreate( &start );
  hipEventCreate( &stop );
  hipEventRecord( start, 0 );


  // printf("\nCUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

  for (int i = 0; i < n_iter; i++)
  {
    hipMalloc((void**)&dev_src, N * sizeof( uint8_t ));
    hipMalloc((void**)&dev_dst, N * sizeof( uint8_t ));
    hipMalloc((void**)&dev_transfer_out, N * sizeof( uint8_t ));
    hipMalloc((void**)&dev_kernel, sizeof( kernel ));

    hipMemcpy( dev_src, src, N * sizeof(uint8_t), hipMemcpyHostToDevice );
    hipMemcpy( dev_kernel, kernel, sizeof(kernel), hipMemcpyHostToDevice );

    int threadsPerBlock = prop.maxThreadsPerBlock;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    if (blocksPerGrid > MAX_BLOCK_PER_GRID)
    {
      blocksPerGrid = MAX_BLOCK_PER_GRID;
    }

    transfer<<< blocksPerGrid, threadsPerBlock >>>( dev_src, dev_transfer_out, 
                                                  height, width, 2000, 2000 );
  
    Convolution2D<<< blocksPerGrid, threadsPerBlock >>>( dev_transfer_out, dev_dst, height, width, 
                                                        dev_kernel, 9 );

    hipMemcpy( dst, dev_dst, N * sizeof(uint8_t), hipMemcpyDeviceToHost );

    hipFree( dev_src );
    hipFree( dev_dst );
    hipFree( dev_transfer_out );
    hipFree( dev_kernel );

  }

  hipEventRecord( stop, 0 );
  hipEventSynchronize( stop );
  hipEventElapsedTime( &gpuTime, start, stop );

  hipEventDestroy( start );
  hipEventDestroy( stop );

  printf( "\nTime for %d X %d X 3 image = %f milliseconds\n", width, height, gpuTime / n_iter );

  int err = stbi_write_jpg( path_to_save, width, height, ch, dst, width*ch );

  if (err == 0)
  {
    printf("\nWrite/Read error\n");
  }

  stbi_image_free( src );
  free( dst );

  return 0;
}